#include <hip/hip_runtime.h>
#include "gemm.h"
#include <iostream>

float gemm(float *A, float *B, float *C, int iter, int opt) {
    float *dev_A, *dev_B, *dev_C;
    hipMalloc((void**)&dev_A, MM*KK*sizeof(float));
    hipMalloc((void**)&dev_B, KK*NN*sizeof(float));
    hipMalloc((void**)&dev_C, MM*NN*sizeof(float));
    hipMemcpy(dev_A, A, sizeof(float)*MM*KK, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, sizeof(float)*KK*NN, hipMemcpyHostToDevice);
    float time_elapsed = 0.0;
    switch(opt) {
        case 0:
            time_elapsed = opt0(dev_A, dev_B, dev_C, iter);
            break;
        case -5:
            time_elapsed = opt05(dev_A, dev_B, dev_C, iter);
            break;
        case 1:
            time_elapsed = opt1(dev_A, dev_B, dev_C, iter);
            break;
        case 2:
            time_elapsed = opt2(dev_A, dev_B, dev_C, iter);
            break;
        default:
            time_elapsed = baseline(dev_A, dev_B, dev_C, iter);
            break;
    }
    hipMemcpy(C, dev_C, sizeof(float) * MM * NN, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    auto err = hipGetLastError();
    if (err) {
        std::cout << "[ERROR] Something error when execuate kernel, err = " << err << std::endl;
    }

    return time_elapsed;
}